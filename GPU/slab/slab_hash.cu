#include "hip/hip_runtime.h"
//
// Created by jing on 2019-03-17.
//

//#include <cstdint>
#include "hip/hip_runtime.h"
#include "slab_hash.h"
#include "include/dy_hash.h"
#include "gputimer.h"
//#include <cstdio>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#define ulong unsigned long long int
#define make_ulong(k,v) ((((ulong)v)<<32)+k)
#define getk_from_ulong(kv) ((unsigned)((kv)&0xffffffff))
#define getv_from_ulong(kv) ((unsigned)((kv)>>32))
#define gtid (blockIdx.x * blockDim.x + threadIdx.x)

#define RAW_PTR(x) thrust::raw_pointer_cast((x).data())

//#define NUM_DATA 100000000
//#define NUM_DATA 500
//#define NUM_DATA 500

#define DE_ACTIVE 0
#define ACTIVE 1
#define TYPE unsigned int




class Node{
public:
    unsigned _entry[32];

    //__device__ Node* next(){ return (Node*)&_entry[30]; }
    __device__ ulong getkv(int pos){
        return ((ulong*)_entry)[pos];
    }
    __device__ ulong getnext(){
        return getkv(15);
    }

    __device__ bool ptrCAS(ulong old_val,ulong node){
        long old=atomicCAS((ulong*)&_entry[30],old_val,node);
        return old==old_val;
    }

    __device__ bool set_kv_on_pos(int k,int v,int pos,ulong oldkv){
        ulong newkv=make_ulong(k,v);
        ulong old=atomicCAS((ulong*)&_entry[pos*2],oldkv,newkv);
#if CAS_debug
        printf(">>>CAS:id:%d,kv %d,%d pos %d \n>>>CAS:oldkv %lld,%lld CASold: %lld,%lld kv: %lld, %lld pos: %d,%d\n",
                threadIdx.x,k,v,pos,
                oldkv>>32,oldkv& 0xffffffff,old>>32,old& 0xffffffff,newkv>>32,newkv& 0xffffffff,
                _entry[pos*2],_entry[pos*2+1]);
        show("in CAS",k);
#endif

        return old==oldkv;
    }

    __device__ bool set_ptr(Node* ptr){
        return ptrCAS(getnext(),(ulong)ptr);
    }

    __device__ void show(const char* src,int pos=0){
        printf("%s: val:%d tid:%d loc:%lx",src,pos,gtid,&_entry[0]);
#if kv_in_Node
        printf("\n");
        for(int i=0;i<30;i++){
            printf("%d ",_entry[i]);
        }
#endif
        printf("next: %lx\n",getnext());
    }
};

class MemPool{
public:
    Node* data[MEM_POOL_NUM];
    unsigned int pos[MEM_POOL_NUM];
    MemPool(){}
    MemPool(int s){
        printf("memory pool %d initing\n",s);
        Node* ptr;
        hipMalloc((void**)&ptr,sizeof(Node)*MEM_POOL_NUM*NODE_OF_POOL);
        hipMemset(ptr,0, sizeof(Node)*MEM_POOL_NUM*NODE_OF_POOL);
        for(int i=0;i<MEM_POOL_NUM;i++){
            data[i]=ptr+i*NODE_OF_POOL;
        }
        memset(pos,0, sizeof(int)*MEM_POOL_NUM);
        printf("memory pool %d inited\n",s);
    }

    __device__ void show_pool(){
        for(int i=0;i<NODE_OF_POOL;i++){
            printf("%d %d %llx\n",0,i,&data[0][i]);
            data[0][i].show("mempool show",i);
        }
    }

    __device__ Node* get_Node(){
#if show_memory
        show_pool();
#endif
        int tid = gtid>>5;
        int old=atomicInc(&pos[tid],NODE_OF_POOL+10);
        if(old>=NODE_OF_POOL)
            return NULL;
#if check_memory_alloc
        printf("get one:%d loc:%llx\n",old,&data[tid][old]);
#endif
        return &data[tid][old];
    }
    __device__ void free_Node(Node* n){
        int tid = gtid>>5;
        int unuse_pos=n-data[tid];
        assert(unuse_pos>=0);
        atomicCAS(&pos[tid],unuse_pos+1,unuse_pos);
    }
};

__device__ MemPool* mem_pool;

void printinit(){
    static int a=1;
    if(a==1){
        printf("slab list inited\n");
        a=0;
    }
}

class SlabList{
public:
    Node* first;
    static int inited;

    SlabList(){
        printinit();
        hipMalloc((void**)&first,sizeof(Node));
        hipMemset(first,0, sizeof(Node));
    }

    __device__ bool alloc_new_node(Node* tmp) {

        int id_in_warp = threadIdx.x & 0x1f;
        if (id_in_warp == 0) {
            Node *nxt = mem_pool->get_Node();
            if (nxt == NULL) {
                printf("!!!! error , pool used over\n");
                printf("!!!! error , pool used over tid:%d ,nxt:%lld \n", gtid,nxt);
                printf("!!!! error , pool used over\n");
                return false;
            }
            if (!tmp->set_ptr(nxt)) {
                mem_pool->free_Node(nxt);
            }

        }
        return true;
    }

    __device__ bool check_target_and_CAS(int k,int v,int &is_active,int target,Node* tmp){
        int id_in_warp=threadIdx.x & 0x1f;
        int ballot=__ballot(tmp->_entry[id_in_warp]==target);
        ballot&=MASK_CHECK_kv;
        //  只要还发现 target
        while(ballot!=0){
            int chosen_simd=__ffs(ballot)-1;
            ulong oldkv=tmp->getkv(chosen_simd/2);
            if(getk_from_ulong(oldkv)==target) {
                if(id_in_warp==chosen_simd) {
                    if (tmp->set_kv_on_pos(k, v, chosen_simd / 2, oldkv)) {
                        is_active = DE_ACTIVE;
                    }
                }
                #if insert_debug
                    if(id_in_warp==0){
                        printf("kv %d,%d target %d isa:%d,ballot:%llx\n",k,v,target,is_active,ballot);
                        tmp->show("check in insert",k);
                    }
                #endif
                is_active = __shfl(is_active, chosen_simd, BUCKET_SIZE);
                if (is_active == DE_ACTIVE) return true;
            }
            ballot=__ballot(tmp->_entry[id_in_warp]==target);
            ballot&=MASK_CHECK_kv;
        }
        return false;
    }

    // after ballot , warp insert
    __device__ bool warp_insert(int k,int v){
        int id_in_warp=threadIdx.x & 0x1f;
        Node* tmp=first;
        assert(tmp!=NULL);

        int is_active=ACTIVE;

        while(tmp!=NULL){
#if insert_debug
            if(id_in_warp==0)  tmp->show("check in main while in insert",k);
#endif

            // check exist
//            if( check_target_and_CAS(k,v,is_active,k,tmp) )
//                return true;

            // check null
            if( check_target_and_CAS(k,v,is_active,0,tmp) )
                return true;

            if(NULL == (Node*)(tmp->getnext())){
                alloc_new_node(tmp);
            }
            tmp=(Node*)(tmp->getnext());
        }

        if(id_in_warp==0)
            printf("need resize\n");
        return false;
    }

    __device__ bool check_target_and_return_value(int k,Node* tmp,int write_pos,TYPE* value){
        int id_in_warp=threadIdx.x & 0x1f;
        int ballot=__ballot(tmp->_entry[id_in_warp]==k);
        ballot&=MASK_CHECK_kv;
        if(ballot!=0){
            int chosen_simd=__ffs(ballot)-1;
            ulong oldkv=tmp->getkv(chosen_simd/2);
            if(id_in_warp==chosen_simd) {
                value[write_pos]=getv_from_ulong(oldkv);
            }
            return true;
        }
        return false;
    }


    __device__ bool warp_search(int k,int write_pos,TYPE* value){
        Node* tmp=first;
        assert(tmp!=NULL);
        while(tmp!=NULL) {
            if(check_target_and_return_value(k,tmp,write_pos,value))
                return true;
            tmp=(Node*)(tmp->getnext());
        }
        return false;
    }

    __device__ void show_list(int n){
        Node* tmp=first;
        for(int i=0;i<n && tmp!=NULL;i++){
            tmp->show("list show",i);
            tmp=(Node*)(tmp->getnext());
        }
    }
}; // class slablist


struct SlabHash{
public:
    SlabList slist[TABLE_NUM];
};


__global__ void
kernel_find_test(SlabHash* ht,TYPE *key,TYPE* value,int size){
    int tid = gtid;
    int id_of_warp = (tid) >> 5;
    int step = (gridDim.x * blockDim.x) >> 5;

    for (; id_of_warp < size; id_of_warp += step) {
        int k=key[id_of_warp];
        ht->slist[k % TABLE_NUM].warp_search(k,id_of_warp, value );
    }
#if show_list_in_kernel
    __syncthreads();
    if (gtid == 0) {
        for (int i = 0; i < TABLE_NUM; i++) {
            printf("listnum:%d\n", i);
            ht->slist[i].show_list(300);
        }
    }
#endif
}

__global__ void
kernel_test(SlabHash* ht,TYPE *key,TYPE* value,int size) {
    int tid = gtid;
    int id_of_warp = (tid) >> 5;
    int step = (gridDim.x * blockDim.x) >> 5;

    for (; id_of_warp < size; id_of_warp += step) {
        int k=key[id_of_warp];
        int v=value[id_of_warp];
        ht->slist[k % TABLE_NUM].warp_insert(k, v);
    }
#if show_list_in_kernel
    __syncthreads();
    if (gtid == 0) {
        for (int i = 0; i < TABLE_NUM; i++) {
            printf("listnum:%d\n", i);
            ht->slist[i].show_list(300);
        }
    }
#endif
}

__global__ void
set_mempool(MemPool* pool){
    mem_pool=pool;
}

using namespace std;


unsigned int* Read_Data(char* filename)
{
//     printf("info:filename:%s\n",filename);
    int size=NUM_DATA;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/twitter.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/tpc-h.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/real_2018/l32.dat")==0)
        size=size/10;

    FILE *fid;
    fid = fopen(filename, "rb");
    unsigned int *pos;
    pos = (unsigned int *)malloc(sizeof(unsigned int)*size);//申请内存空间，大小为n个int长度

    if (fid == NULL)
    {
        printf("the data file is unavailable.\n");
        exit(1);
        return pos;
    }
    fread(pos, sizeof(unsigned int), size, fid);
    fclose(fid);
    return pos;
}


bool init_kv(TYPE *key,TYPE *value,char *filename,int size){

//    TYPE *k;
//    k=Read_Data(filename);

//     GenerateUniqueRandomNumbers(key, pool_size);

    for (int i = 0; i < size; i++) {
//        key[i]=k[i];
        key[i] =(TYPE) i;
//        value[i] =(TYPE) 3 * i + 3 + 1;
        value[i] =(TYPE) i;
//        chck[i]  = 0;
    }
//    for (int i = 0; i < size; i++) {
//        printf("k:%d   v:%d\n",key[i],value[i]);
////        chck[i]  = 0;
//    }
    return true;
}

using namespace thrust;
void check_result(TYPE* key,TYPE* value,TYPE* check,int size){
    int tmp=0;
    for(int i=0;i<size;i++){
        if(value[i]!=check[i]){
            if(tmp++<20) printf("false: %d :k:%d v:%d find:%d\n",i,key[i],value[i],check[i]);
        }
    }
    printf("check result:%d insert ,find %d ,not find %d (%d)\n",size,size-tmp,tmp,tmp*1.0/size);
}

void simple_gpu_test(char *filename){

    int size=NUM_DATA;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/twitter.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/tpc-h.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/GPU_Hash/finally-test/data/real_2018/l32.dat")==0)
        size=size/10;


    // alloc data
    thrust::host_vector<TYPE> key(size+1);
    thrust::host_vector<TYPE> value(size+1);


    //init data
    init_kv(RAW_PTR(key),RAW_PTR(value),filename,size);
    printf("=========init over==========\n");

    // copy to  gpu
    thrust::device_vector<TYPE> dkey(key);
    thrust::device_vector<TYPE> dvalue(value);
    thrust::device_vector<TYPE> dcheck(size+1);

    // 初始化hash表
    SlabHash hash;
    SlabHash* dhash;
    hipMalloc((void**)&dhash,sizeof(SlabHash));
    hipMemcpy(dhash,&hash, sizeof(SlabHash),hipMemcpyHostToDevice);
    // 初始化 pool
    MemPool h_pool(0);
    MemPool *d_pool;
    hipMalloc((void**)&d_pool,sizeof(MemPool));
    hipMemcpy(d_pool,&h_pool, sizeof(MemPool),hipMemcpyHostToDevice);
    set_mempool<<<1,1>>>(d_pool);


    // 插入数据
    GpuTimer timer;
    timer.Start();
    kernel_test<<<512,512>>>(dhash,RAW_PTR(dkey),RAW_PTR(dvalue),size);
    timer.Stop();
    double  diff = timer.Elapsed()*1000000;
    printf("<<<time>>> %.2lf ( %.2f)\n",
           (double) diff, (double) (size) / diff);
    hipDeviceSynchronize();
    hipGetLastError();


    // find 数据
    timer.Start();
    kernel_find_test<<<512,512>>>(dhash,RAW_PTR(dkey),RAW_PTR(dcheck),size);
    timer.Stop();
    diff = timer.Elapsed()*1000000;
    printf("<<<time>>> %.2lf ( %.2f)\n",
           (double) diff, (double) (size) / diff);
    hipDeviceSynchronize();
    hipGetLastError();

    thrust::host_vector<TYPE> check(dcheck);
    check_search_result(RAW_PTR(key),RAW_PTR(value),RAW_PTR(check),size);


}