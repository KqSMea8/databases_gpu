#include "hip/hip_runtime.h"
//
// Created by jing on 2019-03-17.
//

//#include <cstdint>
#include "hip/hip_runtime.h"
#include "slab_hash.h"
#include "include/dy_hash.h"
//#include <cstdio>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#define ulong unsigned long long int
#define make_ulong(k,v) ((((ulong)v)<<32)+k)
#define getk_from_ulong(kv) ((unsigned)((kv)&0xffffffff))
#define getv_from_ulong(kv) ((unsigned)((kv)>>32))
#define gtid (blockIdx.x * blockDim.x + threadIdx.x)

#define RAW_PTR(x) thrust::raw_pointer_cast((x).data())

#define NUM_DATA 100000000

#define DE_ACTIVE 0
#define ACTIVE 1
#define TYPE unsigned int




class Node{
public:
    unsigned _entry[32];

    //__device__ Node* next(){ return (Node*)&_entry[30]; }
    __device__ ulong getkv(int pos){
        return ((ulong*)_entry)[pos];
    }
    __device__ ulong getnext(){
        return getkv(15);
    }

    __device__ bool ptrCAS(ulong old_val,ulong node){
        long old=atomicCAS((ulong*)&_entry[30],old_val,node);
        return old==old_val;
    }

    __device__ bool set_kv_on_pos(int k,int v,int pos,ulong oldkv){
        ulong newkv=make_ulong(k,v);
        ulong old=atomicCAS((ulong*)&_entry[pos*2],oldkv,newkv);
#if CAS_debug
        printf(">>>CAS:id:%d,kv %d,%d pos %d \n>>>CAS:oldkv %lld,%lld CASold: %lld,%lld kv: %lld, %lld pos: %d,%d\n",
                threadIdx.x,k,v,pos,
                oldkv>>32,oldkv& 0xffffffff,old>>32,old& 0xffffffff,newkv>>32,newkv& 0xffffffff,
                _entry[pos*2],_entry[pos*2+1]);
        show("in CAS",k);
#endif

        return old==oldkv;
    }

    __device__ bool set_ptr(Node* ptr){
        return ptrCAS(getnext(),(ulong)ptr);
    }

    __device__ void show(const char* src,int pos=0){
        printf("%s: val:%d tid:%d loc:%lx",src,pos,gtid,&_entry[0]);
#if kv_in_Node
        printf("\n");
        for(int i=0;i<30;i++){
            printf("%d ",_entry[i]);
        }
#endif
        printf("next: %lx\n",getnext());
    }
};

class MemPool{
public:
    Node* data[MEM_POOL_NUM];
    unsigned int pos[MEM_POOL_NUM];
    MemPool(){}
    MemPool(int s){
        printf("memory pool %d initing\n",s);
        Node* ptr;
        hipMalloc((void**)&ptr,sizeof(Node)*MEM_POOL_NUM*NODE_OF_POOL);
        hipMemset(ptr,0, sizeof(Node)*MEM_POOL_NUM*NODE_OF_POOL);
        for(int i=0;i<MEM_POOL_NUM;i++){
            data[i]=ptr+i*NODE_OF_POOL;
        }
        memset(pos,0, sizeof(int)*MEM_POOL_NUM);
        printf("memory pool %d inited\n",s);
    }

    __device__ void show_pool(){
        for(int i=0;i<NODE_OF_POOL;i++){
            printf("%d %d %llx\n",0,i,&data[0][i]);
            data[0][i].show("mempool show",i);
        }
    }

    __device__ Node* get_Node(){
#if show_memory
        show_pool();
#endif
        int tid = gtid>>5;
        int old=atomicInc(&pos[tid],NODE_OF_POOL+10);
        if(old>=NODE_OF_POOL) return NULL;
#if check_memory_alloc
        printf("get one:%d loc:%llx\n",old,&data[tid][old]);
#endif
        return &data[tid][old];
    }
    __device__ void free_Node(Node* n){
        int tid = gtid>>5;
        int unuse_pos=n-data[tid];
        assert(unuse_pos>=0);
        atomicCAS(&pos[tid],unuse_pos+1,unuse_pos);
    }
};

__device__ MemPool* mem_pool;

void printinit(){
    static int a=1;
    if(a==1){
        printf("slab list inited\n");
        a=0;
    }
}

class SlabList{
public:
    Node* first;
    static int inited;

    SlabList(){
        printinit();
        hipMalloc((void**)&first,sizeof(Node));
        hipMemset(first,0, sizeof(Node));
    }

    __device__ bool alloc_new_node(Node* tmp) {

        int id_in_warp = threadIdx.x & 0x1f;
        if (id_in_warp == 0) {
            Node *nxt = mem_pool->get_Node();
            if (nxt == NULL) {
                printf("!!!! error , pool used over\n");
                printf("!!!! error , pool used over tid:%d ,nxt:%lld \n", gtid,nxt);
                printf("!!!! error , pool used over\n");
                return false;
            }
            if (!tmp->set_ptr(nxt)) {
                mem_pool->free_Node(nxt);
            }

        }
        return true;
    }

    __device__ bool check_target_and_CAS(int k,int v,int &is_active,int target,Node* tmp){
        int id_in_warp=threadIdx.x & 0x1f;
        int ballot=__ballot(tmp->_entry[id_in_warp]==target);
        ballot&=MASK_CHECK_kv;
        //  只要还发现 target
        while(ballot!=0){
            int chosen_simd=__ffs(ballot)-1;
            ulong oldkv=tmp->getkv(chosen_simd/2);
            if(getk_from_ulong(oldkv)==target) {
                if(id_in_warp==chosen_simd) {
                    if (tmp->set_kv_on_pos(k, v, chosen_simd / 2, oldkv)) {
                        is_active = DE_ACTIVE;
                    }
                }
                #if insert_debug
                    if(id_in_warp==0){
                        printf("kv %d,%d target %d isa:%d,ballot:%llx\n",k,v,target,is_active,ballot);
                        tmp->show("check in insert",k);
                    }
                #endif
                is_active = __shfl(is_active, chosen_simd, BUCKET_SIZE);
                if (is_active == DE_ACTIVE) return true;
            }
            ballot=__ballot(tmp->_entry[id_in_warp]==target);
            ballot&=MASK_CHECK_kv;
        }
        return false;
    }

    // after ballot , warp insert
    __device__ bool warp_insert(int k,int v){
        int id_in_warp=threadIdx.x & 0x1f;
        Node* tmp=first;
        assert(tmp!=NULL);

        int is_active=ACTIVE;
        int i=0;
        while(tmp!=NULL){
            if(i++ > 5) return false;
#if insert_debug
            if(id_in_warp==0)  tmp->show("check in main while in insert",k);
#endif

            // check exist
//            if( check_target_and_CAS(k,v,is_active,k,tmp) )
//                return true;

            // check null
            if( check_target_and_CAS(k,v,is_active,0,tmp) )
                return true;


            if(NULL == (Node*)(tmp->getnext())){
                alloc_new_node(tmp);
            }
            tmp=(Node*)(tmp->getnext());
        }

        if(id_in_warp==0)
            printf("need resize\n");
        return false;
    }

    __device__ bool check_target_and_return_value(int k,Node* tmp,int write_pos,TYPE* value){
        int id_in_warp=threadIdx.x & 0x1f;
        int ballot=__ballot(tmp->_entry[id_in_warp]==k);
        ballot&=MASK_CHECK_kv;
        if(ballot!=0){
            int chosen_simd=__ffs(ballot)-1;
            ulong oldkv=tmp->getkv(chosen_simd/2);
            if(id_in_warp==chosen_simd) {
                value[write_pos]=getv_from_ulong(oldkv);
            }
            return true;
        }
        return false;
    }


    __device__ bool warp_search(int k,int v,int write_pos,TYPE* value){
        Node* tmp=first;
        assert(tmp!=NULL);
        // find
        while(tmp!=NULL) {
            if( check_target_and_return_value(k,tmp,write_pos,value) )
                return true;
            tmp=(Node*)(tmp->getnext());
        }
        return false;
    }

    __device__ void show_list(int n){
        Node* tmp=first;
        for(int i=0;i<n && tmp!=NULL;i++){
            tmp->show("list show",i);
            tmp=(Node*)(tmp->getnext());
        }
    }
};


struct SlabHash{
public:
    SlabList slist[TABLE_NUM];
};


__global__ void
kernel_search(SlabHash* ht){
    int id_in_block=threadIdx.x ;
    int warp_id_in_block=id_in_block>>5;
    int id_in_warp=id_in_block&0x1f;

//    ht->slist[id_in_block%TABLE_NUM].warp_search(warp_id_in_block*10+1,warp_id_in_block*10+2);

    __syncthreads();

    if(id_in_warp==20 && warp_id_in_block ==0){
        ht->slist[warp_id_in_block%TABLE_NUM].first->show("over",warp_id_in_block);
    }
}

__global__ void
kernel_test(SlabHash* ht,TYPE *key,TYPE* value) {
    int tid = gtid;
    int id_of_warp = (blockIdx.x * blockDim.x + threadIdx.x) >> 5;
    int step = (gridDim.x * blockDim.x) >> 5;
    int elem_num=1000;

    for (; id_of_warp < elem_num; id_of_warp += step) {
        ht->slist[id_of_warp % TABLE_NUM].warp_insert(id_of_warp * 100 + + 1, id_of_warp * 100 + 2 );
    }
    __syncthreads();
    if (gtid == 0) {
        for (int i = 0; i < TABLE_NUM; i++) {
            printf("listnum:%d\n", i);
            ht->slist[i].show_list(50);
        }
    }
}

__global__ void
set_mempool(MemPool* pool){
    mem_pool=pool;
}

using namespace std;


unsigned int* Read_Data(char* filename)
{
//     printf("info:filename:%s\n",filename);
    int size=NUM_DATA;
    if(strcmp(filename,"/home/udms/ly/finally-test/data/twitter.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/finally-test/data/tpc-h.dat")==0)
        size=size/2;
    if(strcmp(filename,"/home/udms/ly/data/real_2018/l32.dat")==0)
        size=size/10;

    FILE *fid;
    fid = fopen(filename, "rb");
    unsigned int *pos;
    pos = (unsigned int *)malloc(sizeof(unsigned int)*size);//申请内存空间，大小为n个int长度

    if (fid == NULL)
    {
        printf("the data file is unavailable.\n");
        exit(1);
        return pos;
    }
    fread(pos, sizeof(unsigned int), size, fid);
    fclose(fid);
    return pos;
}


bool init_kv(TYPE *key,TYPE *value,char *filename,int size){

//    key=Read_Data(filename);

//     GenerateUniqueRandomNumbers(key, pool_size);

    for (int i = 0; i < size; i++) {
         key[i]=(TYPE) 3 * i + 3 + 1;
        value[i] =(TYPE) 3 * i + 3 + 1;
//        chck[i]  = 0;
    }
    return true;
}

using namespace thrust;

void simple_gpu_test(char *filename){
    printf("=========init over==========\n");
    int size=NUM_DATA;
//    if(strcmp(filename,"/home/udms/ly/finally-test/data/twitter.dat")==0)
//        size=size/2;
//    if(strcmp(filename,"/home/udms/ly/finally-test/data/tpc-h.dat")==0)
//        size=size/2;
//    if(strcmp(filename,"/home/udms/ly/data/real_2018/l32.dat")==0)
//        size=size/10;
//

    // alloc data
    thrust::host_vector<TYPE> key(size+1);
    thrust::host_vector<TYPE> value(size+1);


    //init data
    init_kv(RAW_PTR(key),RAW_PTR(value),filename,size);
    printf("=========init over==========\n");

    // copy to  gpu
    thrust::device_vector<TYPE> dkey(key);
    thrust::device_vector<TYPE> dvalue(value);


    SlabHash hash;
    SlabHash* dhash;
    hipMalloc((void**)&dhash,sizeof(SlabHash));
    hipMemcpy(dhash,&hash, sizeof(SlabHash),hipMemcpyHostToDevice);
    MemPool h_pool(0);
    MemPool *d_pool;
    hipMalloc((void**)&d_pool,sizeof(MemPool));
    hipMemcpy(d_pool,&h_pool, sizeof(MemPool),hipMemcpyHostToDevice);
    set_mempool<<<1,1>>>(d_pool);
    kernel_test<<<56,512>>>(dhash,RAW_PTR(dkey),RAW_PTR(dvalue));
//    hipDeviceSynchronize();
    hipGetLastError();
}