#include "hip/hip_runtime.h"
//
// Created by jing on 2018/7/1.
//

// add vscode sup



#include "cuckoo.h"
#include <assert.h>
#include <>
#include "api.h"

// Supported operations
#define ADD (0)
#define DELETE (1)
#define SEARCH (2)
#define debug_num 30
#define single_BUCKET 15629 



/// hash table
__constant__ cuckoo table;
#define  get_table_length(i)  get_table_bucket_length(i)
#define  get_table_bucket_length(i) (table.Lsize[i]/BUCKET_SIZE)
/// Lsize0 is the biggest
#define  Lock_pos(num,hash) ((num) * (get_table_length(0)) + hash)


#define parameter_of_hash_function_a(num) (table.hash_fun[num].x)
#define parameter_of_hash_function_b(num) (table.hash_fun[num].y)


/// hash functiong
__device__ __forceinline__ TYPE
get_next_loc(TYPE k,
             TYPE num_table)
{
    return ( k^ parameter_of_hash_function_a(num_table)
               + parameter_of_hash_function_b(num_table)
           ) % PRIME_uint
           % get_table_length(num_table);
}

/// for debug
__device__ void pbucket(bucket *b,int num,int hash,int t_size)
{
    printf("table.%d,%d/%d \n",num,hash,t_size);
    for(int i=0;i<BUCKET_SIZE;i++){
        if(i%8==0) printf("\n\t");
        printf("%d,%d ",b->key[i],b->value[i]);
    }
    printf("\n");
}




__global__ void
cuckoo_insert(TYPE* key, /// key to insert
              TYPE* value, /// value to insert
              TYPE size, /// insert size
              int* resize) /// insert error?
{
    *resize = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    /// for every k

    /// warp cooperation
    int lan_id = threadIdx.x & 0x1f;
    int warp_num_in_all = tid >> 5;

    TYPE myk, myv;
    TYPE evict_time_of_one_thread = 0;
    int hash;
    TYPE operator_hash_table_num = 0;

    /// using for ballot & CAS
    int tmp=2;
    
    /// first read data
    myk = key[warp_num_in_all];
    myv = value[warp_num_in_all];
    
    while (warp_num_in_all < size) {
  
        ///for re lock , try other table
        operator_hash_table_num++;
        operator_hash_table_num %= TABLE_NUM;
        hash = get_next_loc(myk, operator_hash_table_num);

/// step3.1 lock & un compress  TODO: compress
        ///  lock  ,otherwise revote
        if (lan_id == 0) {
            tmp = atomicCAS(&(table.Lock[Lock_pos(operator_hash_table_num, hash)]), 0, 1);
        }//end if
        tmp=__shfl(tmp, 0);
        if(tmp==1) {
            continue;
        }

        /// bucket
        bucket *b = &(table.table[operator_hash_table_num][hash]);

        tmp = __ballot(b->key[lan_id] == myk);


        if (tmp != 0) { /// update
        
            if(lan_id==__ffs(tmp) - 1) {
               b->value[lan_id] = myv;
            }

            table.Lock[Lock_pos(operator_hash_table_num, hash)] = 0;

            tid += BLOCK_NUM * THREAD_NUM;
            warp_num_in_all = tid >> 5;
            evict_time_of_one_thread = 0;

            myk = key[warp_num_in_all];
            myv = value[warp_num_in_all];

            continue;
        }//end check update

/// step3.3      check null & insert
        tmp = __ballot(b->key[lan_id] == 0);

        if (tmp != 0) {
            if (lan_id == __ffs(tmp) - 1) {
                b->key[lan_id] = myk;
                b->value[lan_id] = myv;
            }// insert

            table.Lock[Lock_pos(operator_hash_table_num, hash)] = 0;

            tid += BLOCK_NUM * THREAD_NUM;
            warp_num_in_all = tid >> 5;
            evict_time_of_one_thread = 0;

            myk = key[warp_num_in_all];
            myv = value[warp_num_in_all];

            continue;
        }/// null insert over



/// step3.4     other,we need  cuckoo evict
        TYPE tmpk=myk,tmpv=myv;
        /// choose pos:lan_id evict ,TODO: choose rand?

        int evict_pos=myk & 0x1f;

        myk = b->key[evict_pos];
        myv = b->value[evict_pos];
        b->key[evict_pos] = tmpk;
        b->value[evict_pos] = tmpv;

        evict_time_of_one_thread++;
        table.Lock[Lock_pos(operator_hash_table_num, hash)] = 0;

        /// when one always get leader , mark rehash
        /// check long chain
        if (evict_time_of_one_thread >= MAX_ITERATOR) {
        
            *resize = 1;
            tid += BLOCK_NUM * THREAD_NUM;
            warp_num_in_all = tid >> 5;
            evict_time_of_one_thread = 0;

            myk = key[warp_num_in_all];
            myv = value[warp_num_in_all];

            continue;
        }

    }//while size
}//cucukoo insert


__global__ void
cuckoo_search(TYPE* key, /// key to s
              TYPE* value, /// value to key
              TYPE size) /// s size
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    /// for every k
#if head_info_debug
    if(tid==0) {
    printf(">>>search kernel:\n>>>size:%d  \n", size);
    printf(">>>s_size:t1:%d, t2:%d, t3:%d, t4:%d\n",
            table.Lsize[0], table.Lsize[1], table.Lsize[2], table.Lsize[3]);
    }
#endif

    int lan_id = threadIdx.x & 0x0000001f;
    int warp_num_in_block = threadIdx.x >> 5;
    volatile  __shared__ int warp[( THREAD_NUM)>>5 ];

    TYPE myk;

    int is_active;
    TYPE work_k = 0;

    /// for search
    int hash;
    int operator_hash_table_num;
    int ballot;
    bucket *b;



    /// ((size+31)>>5)<<5 :keep  a warp to active
    while ( tid < (((size + 31) >> 5) << 5) ) {

        if(tid<size) {
            myk = key[tid];
            is_active = 1;/// mark for work
        }

        /// while have work to do
        while (__any(is_active != 0)) {

            operator_hash_table_num=0;


/// step1   start voting ==================================
            if (is_active != 0)
                warp[warp_num_in_block] = lan_id;
#if search_debug
            if(lan_id==0)
                printf("voting: %d\t",warp[warp_num_in_block] );
#endif
            work_k = myk;

/// step2   broadcast ====================================
            work_k=__shfl(work_k, warp[warp_num_in_block]);


/// step3   find in 5 table ===========================

            /// find null or too long
            for (int i = 0; i < TABLE_NUM; i++) {
                operator_hash_table_num = i;
                hash = get_next_loc(work_k, operator_hash_table_num);
                b=&table.table[operator_hash_table_num][hash];

                ballot=__ballot(b->key[lan_id]==work_k);

                /// find it
                if(ballot!=0){
                    if(lan_id==warp[warp_num_in_block]){
                        value[tid]=b->value[__ffs(ballot)-1];
#if search_debug
                        printf("find %d: %d\n",key[tid],value[tid]);
#endif
                        is_active=0;
                    }
                    break;
                }

            }/// end for

            /// can not find
            if(lan_id==warp[warp_num_in_block]){
                if(is_active==1) value[tid]=2;
                    //printf("cannot find k: %d  ,tid:%d ",myk,tid);
                //pbucket(b,operator_hash_table_num,hash,get_table_length(operator_hash_table_num));
                is_active=0;
            }
        }
        tid += BLOCK_NUM * THREAD_NUM;
    }

}//cuckoo_search


/// del and return value
__global__ void
cuckoo_delete(TYPE* key, /// key to del
              TYPE* value, /// value to return
              TYPE size) /// size
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    /// for every k
#if head_info_debug
    if(tid==0) {
    printf(">>>delete kernel:\n>>>size:%d  \n", size);
    printf(">>>s_size:t1:%d, t2:%d, t3:%d, t4:%d\n",
            table.Lsize[0], table.Lsize[1], table.Lsize[2], table.Lsize[3]);
    }
#endif

    int lan_id = threadIdx.x & 0x0000001f;
    int warp_num_in_block = threadIdx.x >> 5;
    volatile  __shared__ int warp[( THREAD_NUM)>>5 ];

    TYPE myk;

    int is_active;
    TYPE work_k = 0;

    /// for search
    int hash;
    int operator_hash_table_num;
    int ballot;
    bucket *b;


    /// ((size+31)>>5)<<5 :keep  a warp to active
    while ( tid < (((size + 31) >> 5) << 5) ) {

        if(tid<size) {
            myk = key[tid];
            is_active = 1;/// mark for work
        }

        /// while have work to do
        while (__any(is_active != 0)) {

            operator_hash_table_num=0;

/// step1   start voting ==================================
            if (is_active != 0)
                warp[warp_num_in_block] = lan_id;
#if search_debug
            if(lan_id==0)
                printf("voting: %d\t",warp[warp_num_in_block] );
#endif
            work_k = myk;

/// step2   broadcast ====================================
            work_k=__shfl(work_k, warp[warp_num_in_block]);


/// step3   find in 5 table ===========================

            /// find null or too long
            for (int i = 0; i < TABLE_NUM; i++) {
                operator_hash_table_num = i;
                hash = get_next_loc(work_k, operator_hash_table_num);
                b=&table.table[operator_hash_table_num][hash];

                ballot=__ballot(b->key[lan_id]==work_k);

                /// find it
                if(ballot!=0){
                    if(lan_id==warp[warp_num_in_block]){
                        value[tid]=b->value[__ffs(ballot)-1];
#if search_debug
                        printf("find %d: %d\n",key[tid],value[tid]);
#endif
///step3.1   if find, set to zero ===========================
                        b->key[__ffs(ballot)-1]=0;
                        b->value[__ffs(ballot)-1]=0;
                        is_active=0;
                    }
                    break;
                }

            }/// end for

            /// can not find
            if(lan_id==warp[warp_num_in_block]){
                is_active=0;
            }
        }
        tid += BLOCK_NUM * THREAD_NUM;
    }
}//cuckoo_delete



void __global__
cuckoo_resize_up(bucket* old_table, /// new table has been set to table
                 int old_size,
                 TYPE num_table_to_resize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;


    int warp_num_in_all = tid >> 5;
    int lan_id = tid & 0x1f;


    /// take kv to insert
    TYPE key, value;

    int hash;


    /// hold old one bucket to op
    bucket *b;

///step1 取新表  ======================
    bucket *new_table = table.table[num_table_to_resize];

///step2 每个warp处理一个bucket ======================
    old_size /= BUCKET_SIZE;
    while (warp_num_in_all < old_size) {

///step2.1  获取自己的bucket ======================
        b = &old_table[warp_num_in_all];

///step2.2 对bucket中各插入对应的位置======================
        key = b->key[lan_id];
        value = b->value[lan_id];
        if (key != 0) {
            /// how to use tid & hash fun
            hash = get_next_loc(key,  num_table_to_resize);
            new_table[hash].key[lan_id] = key;
            new_table[hash].value[lan_id] = value;
        }

        tid += BLOCK_NUM * THREAD_NUM;
        warp_num_in_all = tid >> 5;
    }

}//cuckoo_resize_up

void __global__
cuckoo_resize_down(bucket* old_table,  /// small
                   int old_size,
                   int num_table_to_resize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

#if head_info_debug
    if(tid==0) {
        printf(">>>down_size kernel: %d->%d\n",old_size,table.Lsize[num_table_to_resize]);
        printf(">>>s_size:t1:%d, t2:%d, t3:%d, t4:%d\n",
               table.Lsize[0], table.Lsize[1], table.Lsize[2], table.Lsize[3]);
    }
#endif

    bucket *b=NULL;
    bucket *des_b=NULL;

    /// take kv to insert
    TYPE key, value;

    /// insert position
    int hash;
    int new_bucket_size = table.Lsize[num_table_to_resize] / BUCKET_SIZE;

    /// warp coopration
    int warp_num_in_all = tid >> 5;
    int warp_num_in_block = threadIdx.x >> 5;
    int lan_id = tid & 0x1f;
    int is_active;
    int ballot;

    /// in block , for voting
    volatile __shared__ int warp[(THREAD_NUM) >> 5];

///step1 置换新表  ======================
    /// 与新表对应的表长已设置好
    bucket *new_table = table.table[num_table_to_resize];
    /// end ,next : old->new

#if down_size_debug
if (tid==0)
    printf("step start \n");
#endif

///step2 每个warp处理2个bucket->一个bucket ======================
    /// 分别将 旧表 tid tid+new_bucket_size 两个bucket插入到新表的 tid bucket中

    /// PROBLEM： 这里默认 new_bucket_size * 2 = old_size (api.cpp line 47)
    /// 方法 部分条件下可将old_size 设置为偶数，这样只有在多次downsize之后才会不符合上述条件

    /// PROBLEM: 将两个bucket映射到一个bucket，在元素较多的情况下势必造成部分
    ///   溢出，除了将溢出部分插入到其他表，我们还需要合理安排两个到一个的映射关系使之高
    ///   效转换。
    /// 方法1. 逐个查询，使用原子add
    /// 方法2. 对空位置和非空kv scan，直接得到相应位置，：需要shared或其他数组支持
    /// 方法3. 首先进行简单插入，然后使用warp通信找到空位置插入

    /// one thread one block
    while (warp_num_in_all < new_bucket_size) {  /// new size is smaller

///step2.1 获取新表的bucket  ======================
        /// warp_num_in_all is hash_value
        des_b = &new_table[warp_num_in_all];
#if down_size_debug
        if (tid==0)
            printf("step2.1 start \n");
#endif

///step2.2  获取第一个旧表的bucket ======================
        /// 读入第一个bucket中kv到变量
        b = &old_table[warp_num_in_all];
        key = b->key[lan_id];
        value = b->value[lan_id];
#if down_size_debug
        if(tid==0){
            printf("old table1\n");
            pbucket(b,0,0,0);
        }
        if(warp_num_in_all==0)
            printf("b1-%d: %d,%d\n",lan_id,key,value);
#endif

        int crose_lan_id=31-lan_id;
        /// 空kv再此读入第二个bucket 交叉读取
        b = &old_table[warp_num_in_all + new_bucket_size];
        if (key == 0) {
            key = b->key[crose_lan_id];
            value = b->value[crose_lan_id];
        }

        ///到这里，第一个bucket全部会被读入后面接着写入，第二个部分还未读入

#if down_size_debug
        if(tid==0){
            printf("old table2\n");
            pbucket(b,0,0,0);
        }
        if(warp_num_in_all==0)
            printf("b1-%d: %d,%d\n",lan_id,key,value);
#endif

///step2.3   将不为空的kv插入新表=====================
        des_b->key[lan_id] = key;
        des_b->value[lan_id] = value;
#if  down_size_debug || down_size_cas_insert_debug
        if(tid==0)
            printf("write\n");
        if(tid==0)
            pbucket(des_b,num_table_to_resize,tid,get_table_length(num_table_to_resize));
#endif

        is_active=0;


///step2.4  读取第二个bucket中未存入的kv ======================
        if (key != b->key[crose_lan_id]  /// 从未写入过
            && b->key[crose_lan_id] !=0)  /// 存在值
        {
            key = b->key[crose_lan_id];
            value = b->value[crose_lan_id];
            is_active = 1;
        }
#if down_size_debug || down_size_cas_insert_debug
        if(warp_num_in_block==0)
            printf("b1-%d: %d,%d - %d\n",lan_id,key,value,is_active);
#endif


///step2.5 对新bucket还有的空位进行插入kv======================
        /// PROBLEM: how about skip step2.5 use step3 directly

        /// 如果空位置比较少会比较快，否则可能使用scan会更快
        /// 如果还有空位
        ballot = __ballot(des_b->key[lan_id] == 0);
#if down_size_debug
        if( tid==0 && ballot == 0 )
            printf("step 2.5 , full\n");
#endif
        while (__any(des_b->key[lan_id] == 0)) {
#if down_size_debug
            if(tid==0)
                printf("step 2.5 \n");
#endif
            if(!__any(is_active==1)) break;
#if down_size_debug
            if(tid==0)
                pbucket(des_b,num_table_to_resize,tid,get_table_length(num_table_to_resize));
#endif
            /// 找出空位
            ballot = __ballot(des_b->key[lan_id] == 0);

            /// use hash as tmp to decrease register
            /// 选一个空位
            hash = __ffs(ballot) - 1;

            /// 选一个线程
            if (is_active == 1)
                warp[warp_num_in_block] = lan_id;

            /// insert
            if (warp[warp_num_in_block] == lan_id) {
                des_b->key[hash] = key;
                des_b->value[hash] = value;
                is_active=0;
            }
        }



///step 3  如果位将第二个表中元素全部插入完成，插入到其他表中======================
#if  down_size_debug || down_size_cas_insert_debug
        if(tid==0)
            printf("after2.5 start3\n");
        if(tid==0)
            pbucket(des_b,num_table_to_resize,tid,get_table_length(num_table_to_resize));
#endif
        /// key value has kv to insert
        TYPE work_k,work_v;
        int operator_hash_table_num=0;
        int lead_thread_num;
#if down_size_cas_insert_debug
        if(warp_num_in_all==0) {
            printf("b1-%d: %d,%d - %d\n",lan_id,key,value,is_active);
        }
#endif

        int times_of_evict=0;

        while (__any(is_active != 0)) {
            /// using logic of  cuckoo_insert  (__global__)
            /// how to reuse the code ?

            /// TODO , check too long evict

            work_k = key;
            work_v = value;

/// step3.1 start voting ==================================
            if (is_active != 0)//&& warp[warp_num_in_block]!=lan_id )
                warp[warp_num_in_block] = lan_id;


            /// leader is lead_thread_num
            lead_thread_num = warp[warp_num_in_block];

            if(lead_thread_num==lan_id) times_of_evict++;

/// step3.2   broadcast ====================================
            work_k = __shfl(work_k, lead_thread_num);
            work_v = __shfl(work_v, lead_thread_num);

/// step3.3   insert to the table. ===========================
            operator_hash_table_num ++;
            /// donot insert to table:num_table_to_resize full
            if (operator_hash_table_num==num_table_to_resize ) {
                operator_hash_table_num++;
            }
            operator_hash_table_num %= TABLE_NUM;
            hash = get_next_loc(work_k,  operator_hash_table_num);

/// step3.4   lock   TODO: compress  ===========================
            /// using ballot as tmp to decrease register
            ///  lock  ,otherwise revote
            if (lan_id == lead_thread_num) {
                /// TODO: different length need to sum ,tmp using double length
                ballot = atomicCAS(&(table.Lock[Lock_pos(operator_hash_table_num,hash)]), 0, 1);
            }//end if
            ballot = __shfl(ballot, lead_thread_num);
            if (ballot == 1)
                continue;

            b = &(table.table[operator_hash_table_num][hash]);

#if down_size_cas_insert_debug
            ballot=__ballot(is_active==1);
            if(warp_num_in_block==0 && lan_id==0){
                printf("\n\nactive ballot:%x kv %d,%d lead_thread_num:%d\n",
                       ballot,work_k,work_v,lead_thread_num );
                pbucket(b,operator_hash_table_num,hash,get_table_length(operator_hash_table_num));
            }
#endif
/// step3.5   check exist & insert
            ballot = __ballot(b->key[lan_id] == work_k);
            if (ballot != 0) { /// update

                if (lan_id == lead_thread_num) {
                    b->value[__ffs(ballot) - 1] = value;
                    is_active = 0;
#if down_size_cas_insert_debug
                    if(warp_num_in_block==0) {
                        printf("exit after insert \n");
                        pbucket(b, operator_hash_table_num, hash, get_table_length(operator_hash_table_num));
                    }
#endif
                    table.Lock[Lock_pos(operator_hash_table_num,hash)] = 0;
                    times_of_evict=0;
                }// end if ,upadte

                continue;
            }//end check update

/// step3.6   check null & insert
            ballot = __ballot(b->key[lan_id] == 0);

#if down_size_cas_insert_debug
            if(warp_num_in_block==0)  printf("%d,",lan_id);
            if(tid==0){
                printf("\n\nnull ballot:%x kv %d,%d lead_thread_num:%d \n",
                       ballot,work_k,work_v,lead_thread_num);
            }
#endif
            if (ballot != 0) {
                /// set kv
                if (lan_id == __ffs(ballot) - 1) {
                    b->key[lan_id] = work_k;
                    b->value[lan_id] = work_v;
                    /// free
                    table.Lock[Lock_pos(operator_hash_table_num,hash)] = 0;
#if down_size_cas_insert_debug
                    if(warp_num_in_block==0) {
                        printf("null after insert \n");
                        pbucket(b, operator_hash_table_num, hash, get_table_length(operator_hash_table_num));
                    }
#endif
                }// insert

                /// mark active false
                if (lan_id == lead_thread_num){
                    times_of_evict=0;
                    is_active = 0;
                }

                continue;
            }/// null insert over


/// step3.7     other,we need  cuckoo evict
            if (lan_id == lead_thread_num){
                key = b->key[lan_id];
                value = b->value[lan_id];
                b->key[lan_id] = work_k;
                b->value[lan_id] = work_v;
#if down_size_cas_insert_debug
                if(warp_num_in_block==0) {
                    printf("evict after insert \n");
                    pbucket(b, operator_hash_table_num, hash, get_table_length(operator_hash_table_num));
                }
#endif
                table.Lock[Lock_pos(operator_hash_table_num,hash)] = 0;

                if(times_of_evict>MAX_ITERATOR){
                    is_active=0;
                    printf("downsizeing can insert %d %d,tid %d ",key,value,tid);
                    times_of_evict=0;
                }


            } // evict
        }

        /// TODO:auto configure ,what should be add to tid
        tid += BLOCK_NUM * THREAD_NUM;
        warp_num_in_all = tid >> 5;
    }

}//cuckoo_resize_down

int choose_block_num(TYPE size);

void GPU_cuckoo_resize_up(int num_table_to_resize,
                          TYPE old_size,
                          bucket* new_table,
                          cuckoo *h_table)
{

    checkCudaErrors(hipGetLastError());
    TYPE new_size=old_size*2;

    ///  set table & size it needed
    bucket* old_table=h_table->table[num_table_to_resize];
    h_table->Lsize[num_table_to_resize]=new_size;
    h_table->table[num_table_to_resize]=new_table;
    hipMemcpyToSymbol(HIP_SYMBOL(table),h_table,sizeof(cuckoo));

    /// TODO: auto configure
    /// kernel Configuration

    dim3 block=choose_block_num(old_size);

    /// kernel launch


    GpuTimer timer;
    timer.Start();

    cuckoo_resize_up<<<block,THREAD_NUM>>>(old_table,old_size,num_table_to_resize);

    timer.Stop();
    double diff = timer.Elapsed()*1000000;
    printf("kernel <<<upsize>>>：the time is %.2lf us, ( %.2f Mops)s\n",
           (double)diff, (double)(new_size) / diff);

}//GPU_cuckoo_resize_up

void GPU_cuckoo_resize_down(int num_table_to_resize,
                            TYPE old_size,
                            bucket* new_table,
                            cuckoo *h_table)
{
    /// bucket to size : << 5
    int new_size=((get_table_bucket_size(num_table_to_resize)+1)/2) << 5;
    //printf("down_size : %d : szie%d->%d.",num_table_to_resize,old_size,new_size);

    ///  set table & size it needed
    bucket* old_table=h_table->table[num_table_to_resize];
    h_table->Lsize[num_table_to_resize]=new_size;
    h_table->table[num_table_to_resize]=new_table;
    hipMemcpyToSymbol(HIP_SYMBOL(table),h_table,sizeof(cuckoo));

    dim3 block=choose_block_num(old_size);

    /// kernel launch
    cuckoo_resize_down<<<block,THREAD_NUM>>>(old_table,old_size,num_table_to_resize);


}//GPU_cuckoo_resize_down





/// show table by key,value
__global__ void show_table() {
    if (blockIdx.x * blockDim.x + threadIdx.x > 0) return;
    /// i is the table num
    for (int i = 0; i < TABLE_NUM; i++) {
        printf("\n\n\ntable:%d\n", i);
        /// j is the bucket num
        for (int j = 0; j < get_table_length(i); j++) {
            printf("bucket:%d\n", j);
            /// t is every slot(one bucket has 32 slot)
            for (int t = 0; t < BUCKET_SIZE; t++) {
                ///  8 slot a line
                if (t % 8 == 0) printf("\n\t\t");
                printf(" %d,%d ", table.table[i][j].key[t], table.table[i][j].value[t]);
            }
            printf("\n");
        }
    }
}

void GPU_show_table(){
    show_table<<<1,1>>>();
}


void gpu_lp_insert(TYPE* key,
                   TYPE* value,
                   TYPE size,
                   int* resize)
{
    dim3 block=choose_block_num(size*32);

    GpuTimer time;
    time.Start();

    cuckoo_insert <<< block, THREAD_NUM >>> (key, value, size, resize);

    time.Stop();
    double diff = time.Elapsed() * 1000000;
    printf("kernel <<<insert>>>：the time is %.2lf us ( %.2f Mops)\n",
           (double) diff, (double) (size) / diff);
}//gpu_lp_insert



void gpu_lp_search(TYPE* key,
                    TYPE* ans,
                    TYPE size){
    dim3 block=choose_block_num(size);

    GpuTimer time;
    time.Start();

    cuckoo_search<<<block,THREAD_NUM>>>(key,ans,size);

    time.Stop();
    double diff = time.Elapsed() * 1000000;
    printf("kernel <<<search>>>：the time is %.2lf us, ( %.2f Mops)s\n",
           (double)diff, (double)(size) / diff);
    //    checkCudaErrors(hipGetLastError());
}

void gpu_lp_delete(TYPE* key,
                   TYPE* ans,
                   TYPE size){
    dim3 block=choose_block_num(size);

    GpuTimer time;
    time.Start();

    cuckoo_delete<<<block,THREAD_NUM>>>(key,ans,size);

    time.Stop();
    double diff = time.Elapsed() * 1000000;
    printf("delete <<<delete>>>：the time is %.2lf us, ( %.2f Mops)s\n",
           (double)diff, (double)(size) / diff);
    //    checkCudaErrors(hipGetLastError());
}

void gpu_lp_set_table(cuckoo *h_table) {
    //printf("seting table\n");
    hipMemcpyToSymbol(HIP_SYMBOL(table),h_table,sizeof(cuckoo));
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}

int choose_block_num(TYPE size){
    unsigned int real_block=(size+THREAD_NUM-1)/THREAD_NUM;
    /// 不能超过 BLOCK_NUM
    int block=real_block>BLOCK_NUM ? BLOCK_NUM : real_block;
    ///
    block=block<1?1:block;
    return block;
}




