#include "hip/hip_runtime.h"
//
// Created by jing on 2018/7/1.
//


#include "cuckoo.h"
#include <assert.h>
#include <>





/// hash function
//inline __device__  unsigned
//hash_function(TTT key, TTT size)
//{
//    return key % _PRIME % size ;
//}

// TODO hash fun
/// table size is dynamic
__device__ TYPE
get_next_loc(TYPE k,TYPE v,TYPE num_table,cuckoo* table){
    num_table%=TABLE_NUM;
    //printf("hash: k:%d size:%d ,hash :%d \n",(( k%_PRIME )+num_table),(table->Lsize[num_table]/BUCKET_SIZE),(( k%_PRIME) +num_table) % (table->Lsize[num_table]/BUCKET_SIZE));
    return (( k%_PRIME )+num_table) % (table->Lsize[num_table]/BUCKET_SIZE);
}


__device__ void pbucket(bucket *b,int num,int hash,int t_size){
    printf("table%d,%d/%d \n",num,hash,t_size);
    for(int i=0;i<BUCKET_SIZE;i++){
        printf("%d,%d ",b->key[i],b->value[i]);
    }
    printf("\n");
}


__global__ void
cuckoo_insert(TTT* key, /// key to insert
              TTT* value, /// value to insert
              TTT size, /// insert size
              TTT* resize, /// insert error?
              cuckoo* table, /// hash table
              TTT table_size) {
    *resize = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    /// for every k
    if(tid>= size) return;

#if    head_info
if(tid==0) {
    printf("\n\ninsert kernel:\nsize:%d ,tsize:%d \n", size, table_size);
    printf("table:%x ,t1:%x,t2:%x,t3:%x,t4:%x,t5:%x\n",
           table, table->table[0], table->table[1], table->table[2], table->table[3], table->table[4]);
}
#endif

    int lan_id = tid & 0x0000001f;
    int wrap_id = tid >> 5;

    while (tid < size) {

        int myk = key[tid];
        int myv = value[tid];
        int is_active = 1;/// mark for work

        int work_k = 0;
        int work_v = 0;

        /// for insert
        int hash;
        int hash_table_num;
        int ballot;
        int old_cas;

#if 0
if(lan_id==0)
        printf("\nstart insert kv:%d %d ,my wrap_id:%d ,lan_id: %d \n", wrap_id, lan_id, myk, myv);
#endif
        /// for voting , TODO size??
        volatile __shared__ int wrap[(NUM_BLOCK * NUM_THREADS)>>5 ];

        /// while have work to do
        while (__any(is_active != 0)) {

            hash_table_num=0;

#if insert_debug
            //printf("lan_id: %d, active:%d \n",lan_id,is_active);
ballot=__ballot(is_active != 0);
if(lan_id==0)
printf("\n\n\nstart insert  ,my wrap_id:%d ,ballot:%x\n" ,wrap_id,ballot);
#endif

/// step1   start voting ==================================
            if (is_active != 0 && wrap[wrap_id]!=lan_id )
                wrap[wrap_id] = lan_id;
#if insert_debug
if(lan_id==0)
    printf("voting: %d   ",wrap[wrap_id] );
#endif

            work_k = myk;
            work_v = myv;
            /// over ======
#if insert_debug
if(lan_id==0)
printf("step 2  ", wrap_id, lan_id, myk, myv,wrap[wrap_id]);
#endif

/// step2   broadcast ====================================
            work_k=__shfl(work_k, wrap[wrap_id]);
            work_v=__shfl(work_v, wrap[wrap_id]);

#if insert_debug
            if(lan_id==0)
                printf("kv: %d ,%d  ",work_k,work_v );
#endif
/// step3   insert to the table ===========================
            hash_table_num = work_k%TABLE_NUM;
            hash = get_next_loc(work_k, work_v, hash_table_num,table);

            /// find null or too long
            for (int i = 0; i < MAX_ITERATOR; i++) {

/// step3.1     TODO:lock and compress
                ///  lock  ,otherwise revote
                if(lan_id==wrap[wrap_id]){
                    old_cas=atomicCAS(&(table->Lock[hash_table_num*(table->Lsize[hash_table_num]/32)+hash]),0,1);
                }
                old_cas=__shfl(old_cas, wrap[wrap_id]);
                if(old_cas==1) break;
//                if(lan_id==0)
//                    printf("work lock:%d / %d \n",hash_table_num*(table->Lsize[hash_table_num]/32)+hash,(table->Lsize[hash_table_num]/32)*TABLE_NUM);
                //assert(0);


                /// block
                bucket *b = &(table->table[hash_table_num][hash]);

                assert(b);
#if 0
if(lan_id==0)
printf("step_3.2 hash:%d  ",hash, wrap_id, lan_id, myk, myv);
#endif

                //printf("lan_id: %d, active:%d , b[key]: %d ,work_k %d \n",lan_id,is_active,b->key[lan_id],work_k);
/// step3.2     check exist & insert
                ballot = __ballot(b->key[lan_id] == work_k);   
                if (ballot != 0) { /// update

                    if(lan_id==wrap[wrap_id]){
#if debug_big_data
                        printf("update: key:%d,v:%d->%d\tt:%d,h:%d\n",myk,b->value[lan_id],myv,hash_table_num,hash);
#endif
                        b->value[lan_id]=myv;

                    }
#if insert_debug
if(lan_id==0)
printf("3.2.1 workK: %d ,work_kv :%d ,vot: %d ,ballot:%d  \n", work_k,work_v,wrap[wrap_id],ballot);
#endif
                    if (lan_id == wrap[wrap_id])
                        is_active = 0;
                    table->Lock[hash_table_num*(table->Lsize[hash_table_num]/32)+hash]=0;
                    break;
                }


                ballot = __ballot(b->key[lan_id] == 0);

#if insert_debug
if(lan_id==0)
printf("3.3 ballot:%x \n ", ballot,wrap_id, lan_id, myk, myv);
if(lan_id==0)
    pbucket(b,hash_table_num,hash,table->Lsize[hash_table_num]/32);
#endif

/// step3.3      check null & insert
                if (ballot != 0) {

#if insert_debug
if(lan_id==0)
printf("3.3.1 my wrap_id:%d,ballot: %x ,ffs:%d\n", wrap_id, ballot,__ffs(ballot)-1,lan_id, myk, myv);
#endif

                    // TODO: choose a lan to insert
                    /// set kv

                    if (lan_id == __ffs(ballot)-1) {
#if debug_big_data
                        printf("insert null: bucket:%d,%d kv:%d,%d\tt:%d,h:%d\n",b->key[lan_id],b->value[lan_id],work_k,work_v,hash_table_num,hash);
#endif
                        b->key[lan_id] = work_k;
                        b->value[lan_id] = work_v;
                    }


                    /// mark active false

                    if (lan_id == wrap[wrap_id])
                        is_active = 0;

                    table->Lock[hash_table_num*(table->Lsize[hash_table_num]/32)+hash]=0;
                    /// insert ok ,
                    break;
                }/// insert

#if insert_debug
if(lan_id==0)
printf("3.4  ", wrap_id, lan_id, myk, myv);
#endif

/// step3.4     other,we need  cuckoo evict
                if(lan_id==wrap[wrap_id]){
#if debug_big_data
                    printf("insert evict: %d,%d -> %d,%d\tt:%d,h:%d\n",b->key[lan_id],b->value[lan_id],work_k,work_v,hash_table_num,hash);
#endif
                    work_k=b->key[lan_id];
                    work_v=b->value[lan_id];
                    b->key[lan_id]=myk;
                    b->value[lan_id]=myv;
                }
                work_k=__shfl(work_k, wrap[wrap_id]);
                work_v=__shfl(work_v, wrap[wrap_id]);
#if insert_debug
if(lan_id==0)
    printf("3.5  work_kv: %d %d\n",work_k,work_v);
#endif

                table->Lock[hash_table_num*(table->Lsize[hash_table_num]/32)+hash]=0;
/// step3.5     keep evicted kv and reinsert
                hash_table_num++;
                hash_table_num%=TABLE_NUM;
                hash = get_next_loc(work_k, work_v,hash_table_num ,table);

            }

        }


        tid += NUM_BLOCK * NUM_THREADS;
    }
}



__global__ void
cuckoo_search(TTT* key, /// key to s
              TTT* value, /// value to key
              TTT size, /// s size
              cuckoo* table) { /// hash table
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    /// for every k
#if head_info
    if(tid==0) {
        printf("\n\nfind kernel\n\nsize:%d\n", size);
        printf("table:%x ,t1:%x,t2:%x,t3:%x,t4:%x,t5:%x\n",
               table, table->table[0], table->table[1], table->table[2], table->table[3], table->table[4]);
    }
#endif
    int lan_id = tid & 0x0000001f;
    int wrap_id = tid >> 5;

    while (tid < size) {

        int myk = key[tid];

        int is_active = 1;/// mark for work

        int work_k = 0;
        int work_v;


        /// for s
        int hash;
        int hash_table_num;
        int ballot;
        bucket *b;
#if head_info
        if(lan_id==0)
            printf("my wrap_id:%d ,lan_id: %d,k:%d \n", wrap_id, lan_id, myk);
#endif
        /// for voting , TODO size??
        volatile __shared__ int wrap[(NUM_BLOCK * NUM_THREADS)>>5 ];

        /// while have work to do
        while (__any(is_active != 0)) {

            hash_table_num=0;

            //printf("lan_id: %d, active:%d \n",lan_id,is_active);
            ballot=__ballot(is_active != 0);
#if search_debug
            if(lan_id==0)
                printf("\n\n1 my wrap_id:%d  work_kv:%d %d ,ballot:%x\n", wrap_id, work_k,work_v,ballot);
#endif

/// step1   start voting ==================================
            if (is_active != 0)
                wrap[wrap_id] = lan_id;
#if search_debug
            if(lan_id==0)
                printf("voting: %d\t",wrap[wrap_id] );
#endif

            work_k = myk;

            /// over ======


/// step2   broadcast ====================================
            work_k=__shfl(work_k, wrap[wrap_id]);


            //printf("lan_id: %d, active:%d  ,work_k %d \n",lan_id,is_active,work_k);
/// step3   find in 5 table ===========================
            hash_table_num = work_k % TABLE_NUM;
            hash = get_next_loc(work_k, work_v, hash_table_num,table);

            /// find null or too long
            for (int i = 0; i < TABLE_NUM; i++) {
                b=&table->table[hash_table_num][hash];
                ballot=__ballot(b->key[lan_id]==work_k);

                /// find it
                if(ballot!=0){
                    if(lan_id==wrap[wrap_id]){
                        value[tid]=b->value[__ffs(ballot)-1];
#if search_debug
                        printf("find %d: %d\n",key[tid],value[tid]);
#endif
                        is_active=0;
                    }
                    break;
                }
                hash_table_num++;
                hash_table_num%=TABLE_NUM;
                hash=get_next_loc(work_k, work_v,hash_table_num,table);
            }
            if(lan_id==wrap[wrap_id]){
                is_active=0;
            }
        }
        tid += NUM_BLOCK * NUM_THREADS;
    }

}


/// TODO: reshah
__global__ void
rehash(TTT* rkey,TTT* rvalue,TTT old_size,TTT table_size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;



    //insert table
    while(tid<old_size){


        tid+=NUM_BLOCK*NUM_THREADS;
    }
}


/// TODO: rehash
void gpu_rehash(TTT old_size,TTT new_table_size){
    //malloc
    printf("----rehash size:  %d --> %d\n",old_size,new_table_size);
    TTT* d_key,*d_value;
    hipMalloc((void**)&d_key, sizeof(TTT)*new_table_size);
    hipMalloc((void**)&d_value, sizeof(TTT)*new_table_size);
    hipMemset(d_key,0, sizeof(TTT)*new_table_size);
    hipMemset(d_value,0, sizeof(TTT)*new_table_size);
    
    checkCudaErrors(hipGetLastError());
    rehash<<<NUM_BLOCK,NUM_THREADS>>>(d_key,d_value,old_size,new_table_size);
    checkCudaErrors(hipGetLastError());


}



/// show table by key,value
__global__ void show_table(cuckoo* table){
    if(blockIdx.x * blockDim.x + threadIdx.x>0 ) return;
    for(int i=0;i<TABLE_NUM;i++){
        printf("\n\n\ntable:%d\n",i);
        for(int j=0;j<(table->Lsize[i])/BUCKET_SIZE;j++){
            for(int t=0;t<BUCKET_SIZE;t++) {
                if(t==16) printf("\n    ");
                printf(" %d,%d ", table->table[i][j].key[t], table->table[i][j].value[t]);
            }
            printf("\n");
        }
    }
}


void gpu_lp_insert(TTT* key,TTT* value,TTT size,TTT* resize,cuckoo *table,TTT &table_size){

    //in main
    // st is you operator num
    unsigned int real_block=((unsigned int)size+NUM_THREADS-1)/NUM_THREADS;
    dim3 block=real_block>NUM_BLOCK ? NUM_BLOCK : real_block;

    //printf("start gpulpi\n");
//    checkCudaErrors(hipGetLastError());
    cuckoo_insert<<<block,NUM_THREADS>>>(key,value,size,resize,table,table_size);
#if __show_table
    show_table<<<1,1>>>(table);
#endif
    int* a=new int[1];
    hipDeviceSynchronize();
//    checkCudaErrors(hipGetLastError());
    hipMemcpy(a,resize,sizeof(TTT),hipMemcpyDeviceToHost);
//    checkCudaErrors(hipGetLastError());
//    if(*a != 0){
//        *a=0;
//        printf("need resize:！！！");
//        int old_size=table_size;
//        table_size *= NUM_grow_ratio;
//        gpu_rehash(old_size,table_size);
//        gpu_lp_insert(key,value,size,resize,table_size);
//    }
//    checkCudaErrors(hipGetLastError());


  
}

//void gpu_lp_delete();

void gpu_lp_search(TTT* key,TTT* ans,TTT size,cuckoo* table){
    unsigned int real_block=(size+NUM_THREADS-1)/NUM_THREADS;
    dim3 block=real_block>NUM_BLOCK ? NUM_BLOCK : real_block;

//    checkCudaErrors(hipGetLastError());
    cuckoo_search<<<block,NUM_THREADS>>>(key,ans,size,table);
    hipDeviceSynchronize();
//    checkCudaErrors(hipGetLastError());

}




